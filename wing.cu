#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GL/glut.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;



void launch_kernel()
{
    // execute the kernel
    dim3 block(8, 8, 1);
    //dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //simple_vbo_kernel<<< grid, block>>>(pos, mesh_width, mesh_height, time);
}

void runCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    // execute the kernel
    //    dim3 block(8, 8, 1);
    //    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

    launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    hipGraphicsUnregisterResource(vbo_res);

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

void display()
{
    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();
}

int main(int argc, char **argv)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());

    // register callbacks
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);

    glutCloseFunc(cleanup);

    // create VBO
    createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

    // run the cuda part
    runCuda(&cuda_vbo_resource);

    // start rendering mainloop
    glutMainLoop();

}